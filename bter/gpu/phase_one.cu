#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void setup_random_kernel(hiprandState *state, unsigned long seed) {
    int idx = blockIdx.x * threadIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void phase_one_shift(double *block_b, double *block_i, double *block_n, int *shift, hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Compute block and compute its offset
    shift[idx] = (int) __double2int_rn(
            block_i[idx] + __double2int_rd(hiprand_uniform(&state[idx]) * block_b[idx]) * block_n[idx]);

}

__global__ void phase_one_i(int *i, double *block_b, double *block_i, double *block_n, int *shift, hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Choose first node
    i[idx] = (int) __double2int_rn(__double2int_rd(hiprand_uniform(&state[idx]) * block_n[idx]) + shift[idx]);

}

__global__ void phase_one_j(int *i, int *j, double *block_b, double *block_i, double *block_n, int *shift,
                            hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Choose second node
    // "Without replacement"
    j[idx] = (int) __double2int_rn(__double2int_rd(hiprand_uniform(&state[idx]) * (block_n[idx] - 1)) + shift[idx]);

    // Remove loops
    if (j[idx] >= i[idx]) {
        ++j[idx];
    }
}

void cuda_wrapper_phase_one(int *i, int *j,
                            double *block_b, double *block_i, double *block_n,
                            int length) {
    hiprandState *devStates;
    hipMalloc(&devStates, length * sizeof(hiprandState));

    int *cuda_i, *cuda_j;

    double *cuda_block_b, *cuda_block_i, *cuda_block_n;

    size_t size_output = length * sizeof(int);
    size_t size_input = length * sizeof(double);

    gpuErrchk(hipMalloc((void **) &cuda_i, size_output));
    gpuErrchk(hipMalloc((void **) &cuda_j, size_output));

    gpuErrchk(hipMalloc((void **) &cuda_block_b, size_input));
    gpuErrchk(hipMalloc((void **) &cuda_block_i, size_input));
    gpuErrchk(hipMalloc((void **) &cuda_j, size_input));

    gpuErrchk(hipMemcpy(cuda_block_b, block_b, size_input, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cuda_block_i, block_i, size_input, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cuda_block_n, block_n, size_input, hipMemcpyHostToDevice));

    // Temporary shift array only on device
    int *cuda_shift;
    gpuErrchk(hipMalloc((void **) &cuda_shift, size_input));

    int blocksize = 256;
    int nblock = length / blocksize + (length % blocksize == 0 ? 0 : 1);
    setup_random_kernel <<< nblock, blocksize >>> (devStates, time(NULL));
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    phase_one_shift << < nblock, blocksize >> > (cuda_block_b, cuda_block_i, cuda_block_n, cuda_shift, devStates);

    phase_one_i << < nblock, blocksize >> > (cuda_i, cuda_block_b, cuda_block_i, cuda_block_n, cuda_shift, devStates);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    phase_one_j << < nblock, blocksize >> >
                             (cuda_i, cuda_j, cuda_block_b, cuda_block_i, cuda_block_n, cuda_shift, devStates);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipMemcpy(i, cuda_i, size_output, hipMemcpyDeviceToHost);
    hipMemcpy(j, cuda_j, size_output, hipMemcpyDeviceToHost);

    // FREE
    hipFree(cuda_shift);

    hipFree(cuda_i);
    hipFree(cuda_j);

    hipFree(cuda_block_b);
    hipFree(cuda_block_i);
    hipFree(cuda_block_n);
}

void cuda_wrapper_phase_two() {

}
