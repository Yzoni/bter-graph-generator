#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void setup_random_kernel(hiprandState *state, unsigned long seed) {
    int idx = blockIdx.x * threadIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void phase_two_fill(double *phase_two_shift_fill, double *phase_two_sz_fill, hiprandState *state) {
    int idx = blockIdx.x * threadIdx.x * blockDim.x;
    phase_two_fill[idx] = phase_two_shift_fill[idx] + floor(hiprand_uniform(&state[idx] * phase_two_sz_fill[idx]);
} 

__global__ void phase_two_bulk(double *phase_two_shift_bulk, double *phase_two_sz_bulk, double *phase_two_bulk, hiprandState *state) {
    int idx = blockIdx.x * threadIdx.x * blockDim.x;
    phase_two_bulk[idx] = phase_two_shift_bulk[idx] + floor(hiprand_uniform(&state[idx]) * phase_two_sz_bulk[idx]); 
}

__global__ void phase_two_d(double *phase_two_fill, double *phase_two_bulk) {
    int idx = blockIdx.x * threadIdx.x * blockDim.x;
    if (hiprand_uniform(&state[idx]) < phase_two_fill[idx]) {
        phase_two[idx] = (int) __double2int_rn(phase_two_fill[idx]) ;
    } else {
        phase_two[idx] = (int) __double2int_rn(phase_two_bulk[idx]); 
    }
}

void cuda_wrapper_phase_two(int phase_two_i, int phase_two_j,
                            int length) {
            
        hiprandState *devStates;
        hipMalloc(&devStates, length * sizeof(hiprandState);        

        int *cuda_i, *cuda_j;

        double *cuda_shift_fill, *cuda_sz_fill; 
        double *cuda_shift_bulk, *cuda_sz_bulk;

        size_t size_output = length * sizeof(int);
        size_t size_input = length * sizeof(double);

        gpuErrchk(hipMalloc((void **) &cuda_i, size_output));
        gpuErrchk(hipMalloc((void **) &cuda_j, size_output));

        gpuErrchk(hipMalloc((void **) &cuda_shift_fill, size_input));
        gpuErrchk(hipMalloc((void **) &cuda_sz_fill, size_input));

        gpuErrchk(hipMalloc((void **) &cuda_shift_bulk, size_input));
        gpuErrchk(hipMalloc((void **) &cuda_sz_bulk, size_input));

        phase_two_fill(phase_two_shift_fill, phase_two_sz_fill, devStates);
        phase_two_bulk(phase_two_shift_bulk, phase_two_sz_bulk, phase_two_bulk, devStates)
       
}
