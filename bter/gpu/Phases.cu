#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include "PhasesKernel.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void cuda_wrapper_phase_one(int *i, int *j,
                            double *block_b, double *block_i, double *block_n,
                            int length) {
    hiprandState *devStates;
    hipMalloc(&devStates, length * sizeof(hiprandState));

    int *cuda_i, *cuda_j;

    double *cuda_block_b, *cuda_block_i, *cuda_block_n;

    size_t size_output = length * sizeof(int);
    size_t size_input = length * sizeof(double);

    gpuErrchk(hipMalloc((void **) &cuda_i, size_output));
    gpuErrchk(hipMalloc((void **) &cuda_j, size_output));

    gpuErrchk(hipMalloc((void **) &cuda_block_b, size_input));
    gpuErrchk(hipMalloc((void **) &cuda_block_i, size_input));
    gpuErrchk(hipMalloc((void **) &cuda_block_n, size_input));

    gpuErrchk(hipMemcpy(cuda_block_b, block_b, size_input, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cuda_block_i, block_i, size_input, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cuda_block_n, block_n, size_input, hipMemcpyHostToDevice));

    // Temporary shift array only on device
    int *cuda_shift;
    gpuErrchk(hipMalloc((void **) &cuda_shift, size_input));

    int blocksize = 256;
    int nblock = length / blocksize + (length % blocksize == 0 ? 0 : 1);

    // Shift
    setup_random_kernel << < nblock, blocksize >> > (devStates, time(NULL), length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    phase_one_shift << < nblock, blocksize >> >
                                 (cuda_block_b, cuda_block_i, cuda_block_n, cuda_shift, devStates, length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // i
    setup_random_kernel << < nblock, blocksize >> > (devStates, time(NULL), length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    phase_one_i << < nblock, blocksize >> >
                             (cuda_i, cuda_block_b, cuda_block_i, cuda_block_n, cuda_shift, devStates, length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // j
    setup_random_kernel << < nblock, blocksize >> > (devStates, time(NULL), length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    phase_one_j << < nblock, blocksize >> >
                             (cuda_i, cuda_j, cuda_block_b, cuda_block_i, cuda_block_n, cuda_shift, devStates, length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipMemcpy(i, cuda_i, size_output, hipMemcpyDeviceToHost);
    hipMemcpy(j, cuda_j, size_output, hipMemcpyDeviceToHost);

    // FREE
    hipFree(cuda_shift);

    hipFree(cuda_i);
    hipFree(cuda_j);

    hipFree(cuda_block_b);
    hipFree(cuda_block_i);
    hipFree(cuda_block_n);
}

void cuda_wrapper_phase_two(double *phase_two_shift_fill, double *phase_two_sz_fill,
                            double *phase_two_shift_bulk, double *phase_two_sz_bulk,
                            int *phase_two,
                            int length) {

    hiprandState *devStates;
    hipMalloc(&devStates, length * sizeof(hiprandState));

    int *cuda_phase_two;

    double *cuda_shift_fill, *cuda_sz_fill;
    double *cuda_shift_bulk, *cuda_sz_bulk;

    size_t size_output = length * sizeof(int);
    size_t size_input = length * sizeof(double);

    gpuErrchk(hipMalloc((void **) &cuda_phase_two, size_output));

    gpuErrchk(hipMalloc((void **) &cuda_shift_fill, size_input));
    gpuErrchk(hipMalloc((void **) &cuda_sz_fill, size_input));

    gpuErrchk(hipMalloc((void **) &cuda_shift_bulk, size_input));
    gpuErrchk(hipMalloc((void **) &cuda_sz_bulk, size_input));

    double *cuda_fill, *cuda_bulk;
    gpuErrchk(hipMalloc((void **) &cuda_fill, size_input));
    gpuErrchk(hipMalloc((void **) &cuda_bulk, size_input));

    gpuErrchk(hipMemcpy(cuda_shift_fill, phase_two_shift_fill, size_input, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cuda_sz_fill, phase_two_sz_fill, size_input, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(cuda_shift_bulk, phase_two_shift_bulk, size_input, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cuda_sz_bulk, phase_two_sz_bulk, size_input, hipMemcpyHostToDevice));

    int blocksize = 256;
    int nblock = length / blocksize + (length % blocksize == 0 ? 0 : 1);

    setup_random_kernel << < nblock, blocksize >> > (devStates, time(NULL), length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    phase_two_fill << < nblock, blocksize >> > (cuda_shift_fill, cuda_sz_fill, cuda_fill,
            devStates, length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    setup_random_kernel << < nblock, blocksize >> > (devStates, time(NULL), length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    phase_two_bulk << < nblock, blocksize >> > (cuda_shift_bulk, cuda_sz_bulk, cuda_bulk,
            devStates, length);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // COPY BACK
    gpuErrchk(hipMemcpy(phase_two, cuda_phase_two, size_output, hipMemcpyDeviceToHost));

    // FREE
    hipFree(cuda_shift_fill);
    hipFree(cuda_sz_fill);
    hipFree(cuda_shift_bulk);
    hipFree(cuda_sz_bulk);

    hipFree(cuda_fill);
    hipFree(cuda_bulk);

    hipFree(cuda_phase_two);
}
